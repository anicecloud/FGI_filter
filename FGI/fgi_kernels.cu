#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>
#include <thrust/fill.h>
#include <thrust/functional.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>
#include <thrust/device_vector.h>
#include <chrono>
#include "tmwtypes.h"
#include "cxxopts.hpp"
#include "fgi_device_functions.h"

__global__ void computeImpulsivity(const double* imageIn, double* impulsivityDegrees, uint32_T rows, uint32_T cols,  uint32_T cuts, uint32_T roadElements, uint32_T windowSize){
	
	uint32_T x =  blockIdx.x * blockDim.x + threadIdx.x; 
	uint32_T y = blockIdx.y * blockDim.y + threadIdx.y; 
	uint32_T index = x+y*cols*rows;
	
	if(x < cols*rows && y < cuts){
		uint8_T maxRange = (2*windowSize+1)*(2*windowSize+1);
		uint8_T ws = maxRange;
		uint32_T windowIndexes[9];
		double distances[9];
		calcWindowIndexes(windowIndexes, ws, rows, cols, windowSize, (2*windowSize+1), x );
		ws = min(ws, maxRange);
		double road_m = 0;
		uint32_T offset = y*cols*rows;
		
		uint32_T elements = min(ws, roadElements);
		road_m = calculateRoad(windowIndexes, ws, imageIn, index, elements, distances, offset);

		if(road_m <= P1) {
			impulsivityDegrees[index] = 0;
		} else if(road_m >= P2) {
			impulsivityDegrees[index] = 1;
		} else {
			impulsivityDegrees[index] = (road_m - P1) / (P2 - P1);
		}
	}
}


__global__ void fuzzyFilter(double* imageIn, uint32_T windowSize, const double *impulsivityDegrees, uint32_T rows, uint32_T cols, uint32_T cuts, uint32_T q) {
    uint32_T x =  blockIdx.x * blockDim.x + threadIdx.x; 
	uint32_T y = blockIdx.y * blockDim.y + threadIdx.y; 
	uint32_T index = x+y*cols*rows;
	
	if(x < cols*rows && y < cuts){
		uint8_T maxRange = (2*windowSize+1)*(2*windowSize+1);
		double oldPx = imageIn[index], newPx;
		uint32_T offset = y*cols*rows;

		uint8_T ws = maxRange;
		uint32_T windowIndexes[9];
		double dist[9];
		calcWindowIndexes(windowIndexes, ws, rows, cols, windowSize, (2*windowSize+1), x );
		for(uint i = 0; i < ws; i++){
				dist[i]=fabs(oldPx-imageIn[windowIndexes[i]+offset]);
		}
		bubbleSort(dist, windowIndexes, ws);

		uint32_T localq = min(ws,q);
		double numerator = 0;
		double totalWeights = 0;

		for(uint32_T i = 0; i < localq; i++) {
			double pixel = imageIn[windowIndexes[i]+offset];
			double weight = calculateWeight(index, oldPx, windowIndexes[i]+offset, pixel, impulsivityDegrees);
			numerator += pixel * weight;
			totalWeights += weight;
		}
		
		newPx =  numerator / totalWeights;
		imageIn[index] = newPx;
	}
}
